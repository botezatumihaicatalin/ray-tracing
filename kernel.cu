#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include "utils/cuda_scoped_ptr.hpp"
#include "utils/cuda_shared_ptr.hpp"
#include "rendering/Scene.h"
#include <ctime>
#include "cimg/CImg.h"
#include <memory>

__global__ void addKernel(int *c, const int *a, const int *b)
{
  int i = threadIdx.x;
  c[i] = a[i] + b[i];
}


// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
  cuda_scoped_ptr<int> d_a(size);
  cuda_scoped_ptr<int> d_b(size);
  cuda_scoped_ptr<int> d_c(size);

  hipMemcpy(d_a.get(), a, size * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_b.get(), b, size * sizeof(int), hipMemcpyHostToDevice);

  addKernel <<<1, size >>>(d_c.get(), d_a.get(), d_b.get());

  hipDeviceSynchronize();

  hipMemcpy(c, d_c.get(), size * sizeof(int), hipMemcpyDeviceToHost);

  return hipSuccess;
}

int main() {

  Scene scene(640, 480);
  scene.antialiasing(false);
  clock_t t0 = clock();
  std::unique_ptr<glm::vec3[]> pixels(scene.render());
  clock_t t1 = clock();

  printf("Render = %f secs\n", float(t1 - t0) / 1000);

  cimg_library::CImg<float> image(scene.width(), scene.height(), 1, 3, 0);

  uint32_t i = 0;
  for (size_t x = 0; x < scene.width(); x++) {
    for (size_t y = 0; y < scene.height(); y++, i++) {
      for (size_t c = 0; c < 3; c ++) {
        image(x, y, 0, c) = pixels[i][c];
      }
    }
  }

  cimg_library::CImgDisplay main_disp(image, "Click a point");
  while (!main_disp.is_closed()) {
    main_disp.wait();
  }

  return 0;
}
