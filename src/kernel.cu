#define GLM_FORCE_CUDA
#define CUDA_VERSION 8000

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <memory>
#include <ctime>

#include "rendering/Scene.h"
#include <cimg/CImg.h>

#include <tbb/tbb.h>
#include <tbb/tick_count.h>

void load_pixels(cimg_library::CImg<uint8_t>& image, glm::vec3* pixels_buf) {
  if (image.spectrum() != 3) {
    throw std::runtime_error("Can't copy");
  }

  size_t pixels_size = image.width() * image.height();

  tbb::parallel_for(tbb::blocked_range<size_t>(0, pixels_size), 
                    [&](const tbb::blocked_range<size_t>& range) {
    for (size_t idx = range.begin(), idx_end = range.end(); idx < idx_end; idx++) {
      size_t y = idx / image.width(), x = idx % image.width();
      for (size_t c = 0; c < 3; c++) {
        image(x, y, 0, c) = uint8_t(pixels_buf[idx][c] * 255);
      }
    }
  });
}

int main() {

  // Hack, because first malloc is very slow.
  int * t;
  hipMalloc(&t, sizeof(int) * 1000);
  hipFree(t);

  Scene scene(1280, 768);
  scene.antialiasing(true);
  cimg_library::CImg<uint8_t> image(scene.width(), scene.height(), 1, 3, 0);

  tbb::task_scheduler_init init(1);

  double render_avg_s = 0;
  double overall_avg_s = 0;
  size_t measures = 0;

  tbb::tick_count t_start, t_stop1, t_stop2;

  cimg_library::CImgDisplay main_disp(image, "W, A, S, D to move camera and E, R to rotate camera");
  while (!main_disp.is_closed()) {
    
    t_start = tbb::tick_count::now();
    std::unique_ptr<glm::vec3[]> pixels(scene.render());
    t_stop1 = tbb::tick_count::now();
    load_pixels(image, pixels.get());
    t_stop2 = tbb::tick_count::now();

    double duration1 = (t_stop1 - t_start).seconds();
    double duration2 = (t_stop2 - t_start).seconds();

    render_avg_s = (render_avg_s * measures + duration1) / (measures + 1);
    overall_avg_s = (overall_avg_s * measures + duration2) / (measures + 1);

    measures = measures + 1;

    printf("Render = %lf secs\n", render_avg_s);
    printf("Overall = %lf secs\n", overall_avg_s);

    image.display(main_disp);

    if (main_disp.is_keyW()) {
      scene.camera().move_forward(0.3f);
    }

    if (main_disp.is_keyS()) {
      scene.camera().move_backward(0.3f);
    }

    if (main_disp.is_keyA()) {
      scene.camera().move_left(0.3f);
    }

    if (main_disp.is_keyD()) {
      scene.camera().move_right(0.3f);
    }

    if (main_disp.is_keyR()) {
      scene.camera().rotate(0.1f);
    }

    if (main_disp.is_keyE()) {
      scene.camera().rotate(-0.1f);
    }

    if (main_disp.is_keyP()) {
      scene.antialiasing(!scene.antialiasing());
    }

    main_disp.wait();
  }

  return 0;
}
