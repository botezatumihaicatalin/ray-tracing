#include "hip/hip_runtime.h"
#define GLM_FORCE_CUDA
#define CUDA_VERSION 8000

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <memory>
#include <ctime>

#include "rendering/Scene.h"
#include "cimg/CImg.h"

#include "tbb/tbb.h"

void load_pixels(cimg_library::CImg<uint8_t>& image, glm::vec3* pixels_buf) {
  if (image.spectrum() != 3) {
    throw std::runtime_error("Can't copy");
  }

  size_t pixels_size = image.width() * image.height();

  tbb::parallel_for(tbb::blocked_range<size_t>(0, pixels_size), 
                    [&](const tbb::blocked_range<size_t>& range) {
    for (size_t idx = range.begin(), idx_end = range.end(); idx < idx_end; idx++) {
      size_t y = idx / image.width(), x = idx % image.width();
      for (size_t c = 0; c < 3; c++) {
        image(x, y, 0, c) = uint8_t(pixels_buf[idx][c] * 255);
      }
    }
  });
}

int main() {

  Scene scene(800, 600);
  scene.antialiasing(true);
  cimg_library::CImg<uint8_t> image(scene.width(), scene.height(), 1, 3, 0);

  cimg_library::CImgDisplay main_disp(image, "W, A, S, D to move camera and E, R to rotate camera");
  while (!main_disp.is_closed()) {
    
    
    clock_t t0 = clock();
    std::unique_ptr<glm::vec3[]> pixels(scene.render());
    load_pixels(image, pixels.get());
    clock_t t1 = clock();
    printf("Render = %f secs\n", float(t1 - t0) / 1000);

    image.display(main_disp);

    if (main_disp.is_keyW()) {
      scene.camera().move_forward(0.3f);
    }

    if (main_disp.is_keyS()) {
      scene.camera().move_backward(0.3f);
    }

    if (main_disp.is_keyA()) {
      scene.camera().move_left(0.3f);
    }

    if (main_disp.is_keyD()) {
      scene.camera().move_right(0.3f);
    }

    if (main_disp.is_keyR()) {
      scene.camera().rotate(0.1f);
    }

    if (main_disp.is_keyE()) {
      scene.camera().rotate(-0.1f);
    }

    main_disp.wait();
  }

  return 0;
}
